#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N)
        C[i] = A[i] + B[i];
}

const int N = 10;

int main() {
    float h_A[N], h_B[N], h_C[N];
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f * i;
        h_B[i] = 10.0f + i;
    }

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < N; i++) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
